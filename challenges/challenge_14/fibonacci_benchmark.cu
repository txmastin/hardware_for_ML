
#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

#define N (1 << 20)

void fibonacci_cpu(std::vector<unsigned long long>& fib, int n) {
    fib[0] = 0;
    fib[1] = 1;
    for (int i = 2; i < n; ++i) {
        fib[i] = fib[i - 1] + fib[i - 2];
    }
}

__global__ void fibonacci_gpu_linear(unsigned long long* fib, int n) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        fib[0] = 0;
        fib[1] = 1;
        for (int i = 2; i < n; ++i) {
            fib[i] = fib[i - 1] + fib[i - 2];
        }
    }
}

__device__ void mat_mult(unsigned long long a[2][2], unsigned long long b[2][2], unsigned long long res[2][2]) {
    res[0][0] = a[0][0]*b[0][0] + a[0][1]*b[1][0];
    res[0][1] = a[0][0]*b[0][1] + a[0][1]*b[1][1];
    res[1][0] = a[1][0]*b[0][0] + a[1][1]*b[1][0];
    res[1][1] = a[1][0]*b[0][1] + a[1][1]*b[1][1];
}

__device__ void mat_pow(unsigned long long base[2][2], int n, unsigned long long res[2][2]) {
    res[0][0] = 1; res[0][1] = 0;
    res[1][0] = 0; res[1][1] = 1;

    unsigned long long temp[2][2];
    while (n > 0) {
        if (n & 1) {
            mat_mult(res, base, temp);
            res[0][0] = temp[0][0]; res[0][1] = temp[0][1];
            res[1][0] = temp[1][0]; res[1][1] = temp[1][1];
        }
        mat_mult(base, base, temp);
        base[0][0] = temp[0][0]; base[0][1] = temp[0][1];
        base[1][0] = temp[1][0]; base[1][1] = temp[1][1];
        n >>= 1;
    }
}

__global__ void fibonacci_gpu_parallel(unsigned long long* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    if (idx == 0) {
        out[0] = 0;
        return;
    }
    if (idx == 1) {
        out[1] = 1;
        return;
    }

    unsigned long long base[2][2] = {{1, 1}, {1, 0}};
    unsigned long long res[2][2];
    mat_pow(base, idx - 1, res);
    out[idx] = res[0][0];
}

void check_correctness(const std::vector<unsigned long long>& a, const std::vector<unsigned long long>& b) {
    for (int i = 0; i < 100; ++i) {
        if (a[i] != b[i]) {
            std::cerr << "Mismatch at index " << i << ": CPU=" << a[i] << ", GPU=" << b[i] << "\n";
            return;
        }
    }
    std::cout << "Results match (first 100 elements).\n";
}

int main() {
    std::vector<unsigned long long> fib_cpu(N);

    // ---------------- CPU ----------------
    auto start_cpu = std::chrono::high_resolution_clock::now();
    fibonacci_cpu(fib_cpu, N);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::cout << "CPU Time: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(end_cpu - start_cpu).count()
              << " ms\n";

    // ---------------- GPU Linear ----------------
    unsigned long long* d_fib;
    hipMalloc(&d_fib, N * sizeof(unsigned long long));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    fibonacci_gpu_linear<<<1, 1>>>(d_fib, N);
    hipDeviceSynchronize();  
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
      std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms_linear = 0;
    hipEventElapsedTime(&ms_linear, start, stop);
    std::cout << "GPU Linear Time: " << ms_linear << " ms\n";

    std::vector<unsigned long long> fib_gpu_linear(N);
    hipMemcpy(fib_gpu_linear.data(), d_fib, N * sizeof(unsigned long long), hipMemcpyDeviceToHost);
    check_correctness(fib_cpu, fib_gpu_linear);

    // ---------------- GPU Parallel Matrix ----------------
    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    hipEventRecord(start);
    fibonacci_gpu_parallel<<<blocks, threads>>>(d_fib, N);
    hipDeviceSynchronize();  
    err = hipGetLastError();
    if (err != hipSuccess)
      std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms_parallel = 0;
    hipEventElapsedTime(&ms_parallel, start, stop);
    std::cout << "GPU Parallel Matrix Time: " << ms_parallel << " ms\n";

    std::vector<unsigned long long> fib_gpu_parallel(N);
    hipMemcpy(fib_gpu_parallel.data(), d_fib, N * sizeof(unsigned long long), hipMemcpyDeviceToHost);
    check_correctness(fib_cpu, fib_gpu_parallel);

    // Cleanup
    hipFree(d_fib);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
