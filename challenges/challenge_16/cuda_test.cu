
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <time.h>

#define BATCH 100
#define IN 100
#define HIDDEN 128
#define OUT 10

__global__ void relu_kernel(float* x, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        x[i] = fmaxf(x[i], 0.0f);
}

void check(hipError_t status, const char* msg) {
    if (status != hipSuccess) {
        fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(status));
        exit(EXIT_FAILURE);
    }
}

int main() {
    float *input, *w1, *w2, *hidden, *output;
    float *d_input, *d_w1, *d_w2, *d_hidden, *d_output;
    hipblasHandle_t handle;
    hipEvent_t start, stop;

    input = (float*)malloc(BATCH * IN * sizeof(float));
    w1 = (float*)malloc(IN * HIDDEN * sizeof(float));
    w2 = (float*)malloc(HIDDEN * OUT * sizeof(float));
    hidden = (float*)malloc(BATCH * HIDDEN * sizeof(float));
    output = (float*)malloc(BATCH * OUT * sizeof(float));

    for (int i = 0; i < BATCH * IN; i++) input[i] = 1.0f;
    for (int i = 0; i < IN * HIDDEN; i++) w1[i] = 0.01f;
    for (int i = 0; i < HIDDEN * OUT; i++) w2[i] = 0.01f;

    check(hipMalloc(&d_input, BATCH * IN * sizeof(float)), "hipMalloc input");
    check(hipMalloc(&d_w1, IN * HIDDEN * sizeof(float)), "hipMalloc w1");
    check(hipMalloc(&d_w2, HIDDEN * OUT * sizeof(float)), "hipMalloc w2");
    check(hipMalloc(&d_hidden, BATCH * HIDDEN * sizeof(float)), "hipMalloc hidden");
    check(hipMalloc(&d_output, BATCH * OUT * sizeof(float)), "hipMalloc output");

    check(hipMemcpy(d_input, input, BATCH * IN * sizeof(float), hipMemcpyHostToDevice), "copy input");
    check(hipMemcpy(d_w1, w1, IN * HIDDEN * sizeof(float), hipMemcpyHostToDevice), "copy w1");
    check(hipMemcpy(d_w2, w2, HIDDEN * OUT * sizeof(float), hipMemcpyHostToDevice), "copy w2");

    hipblasCreate(&handle);
    float alpha = 1.0f, beta = 0.0f;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Forward 1: input @ w1
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, HIDDEN, BATCH, IN,
                &alpha, d_w1, HIDDEN, d_input, IN, &beta, d_hidden, HIDDEN);
    relu_kernel<<<(BATCH*HIDDEN + 255)/256, 256>>>(d_hidden, BATCH * HIDDEN);

    // Forward 2: hidden @ w2
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, OUT, BATCH, HIDDEN,
                &alpha, d_w2, OUT, d_hidden, HIDDEN, &beta, d_output, OUT);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("CUDA Time: %.4f ms\n", ms);

    hipFree(d_input); hipFree(d_w1); hipFree(d_w2);
    hipFree(d_hidden); hipFree(d_output);
    free(input); free(w1); free(w2); free(hidden); free(output);
    hipblasDestroy(handle);
    return 0;
}
